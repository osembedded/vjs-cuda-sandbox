#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/types.h>
#include <unistd.h>
#include <fcntl.h>
#include <stdbool.h>
#include <string.h>

#include "types.h"
#include "tsc.h"

// Turn this switch on if you want to 
// use cuda based acceleration...
#define USE_CUDA

// PPM Edge Enhancement Code
UINT8 *header;
UINT8 *h_R;
UINT8 *h_G;
UINT8 *h_B;
UINT8 *d_R;
UINT8 *d_G;
UINT8 *d_B;
UINT8 *infile;
UINT8 *outfile;
UINT8 *frame_times;

#define PARAMS_GOOD                               \
   (NULL != header &&                             \
    NULL != h_R &&                                \
    NULL != h_G &&                                \
    NULL != h_B &&                                \
    NULL != d_R &&                                \
    NULL != d_B &&                                \
    NULL != d_G &&                                \
    NULL != infile &&                             \
    NULL != outfile &&                            \
    NULL != frame_times)

#ifdef USE_CUDA
#define FREE_MEM                                      \
   free(header);                                      \
   free(h_R);                                         \
   free(h_G);                                         \
   free(h_B);                                         \
   hipFree(d_R);                                     \
   hipFree(d_G);                                     \
   hipFree(d_B);                                     \
   free(infile);                                      \
   free(outfile);                                     \
   free(frame_times);
#else
#define FREE_MEM                                      \
   free(header);                                      \
   free(h_R);                                         \
   free(h_G);                                         \
   free(h_B);                                         \
   free(d_R);                                         \
   free(d_G);                                         \
   free(d_B);                                         \
   free(infile);                                      \
   free(outfile);                                     \
   free(frame_times);
#endif // USE_CUDA


/* User specified */
static char infile_pattern[128];
static char outfile_pattern[128];

void save_ppm_header (int fd, int header_len)
{
   int bytesRead = 0;
   int bytesLeft = 0;
   
   if( -1 == fd )
   {
      printf("Invalid File pointer passed in! Exiting!\n");
      exit (-1);
   }

   bytesLeft = header_len;

   do
   {
      //printf("bytesRead=%d, bytesLeft=%d\n", bytesRead, bytesLeft);
      bytesRead=read(fd, (void *)header, bytesLeft);
      bytesLeft -= bytesRead;
   } while(bytesLeft > 0);
   
   header[header_len]='\0';
   
//    printf("header = %s\n", header);
}

bool interleave_components(UINT8 *ofile, int num_pix, 
                           UINT8 *RR, UINT8 *GG, UINT8 *BB)
{
   int retval = false;
   int ii = 0, jj = 0;

   if(NULL != ofile &&
      NULL != RR &&
      NULL != GG &&
      NULL != BB)
   {
      for(ii = 0; ii < num_pix; ii++)
      {
         // This is where it seg faults if we mess up the memory access...
         ofile[jj++] = RR[ii];
         ofile[jj++] = GG[ii];
         ofile[jj++] = BB[ii];
      }

      retval = true;
   }

   return retval;
}

void write_output_to_file(int fdout, int num_pixels, int header_len,
                          UINT8 *RR, UINT8 *GG, UINT8 *BB)
{
   if( -1 == fdout )
   {
      printf("Invalid File pointer passed in! Exiting!\n");
      exit (-1);
   }

   if( NULL == RR ||
       NULL == GG ||
       NULL == BB )
   {
      printf("NULL parameters passed in! exiting!\n");
      exit (-1);
   }

   write(fdout, (void *)header, header_len);
   
   if( true == interleave_components(outfile, num_pixels, 
                                     RR, GG, BB))
   {
      write(fdout, (void *)outfile, num_pixels*3);
   }
}

bool separate_components (UINT8 *ifile, int num_pix, 
                          UINT8 *RR, UINT8 *GG, UINT8 *BB)
{
   int retval = false;
   int ii = 0, jj = 0;

   if(NULL != ifile &&
      NULL != RR &&
      NULL != GG &&
      NULL != BB)
   {
      for(ii = 0; ii < num_pix; ii++)
      {
         h_R[ii] = ifile[jj++];
         h_G[ii] = ifile[jj++];
         h_B[ii] = ifile[jj++];         
      }

      retval = true;
   }

   return retval;
}

void read_input_from_file(int fdin, int num_pixels, int header_len,
                          UINT8 *RR, UINT8 *GG, UINT8 *BB)
{
   if( -1 == fdin )
   {
      printf("Invalid File pointer passed in! Exiting!\n");
      exit (-1);
   }

   if( NULL == RR ||
       NULL == GG ||
       NULL == BB )
   {
      printf("NULL parameters passed in! exiting!\n");
      exit (-1);
   }

   save_ppm_header(fdin, header_len);

   read(fdin, (void *)infile, num_pixels*3);

   separate_components(infile, num_pixels, 
                       RR, GG, BB);
}

bool open_files(int num, int* fdin, int* fdout)
{
   char in_file[256], out_file[256];

   if(NULL == fdin ||
      NULL == fdout)
   {
      printf("Null parameters passed in in open_files!\n");
      return false;
   }

   snprintf((char *)&in_file[0], 256, infile_pattern, num);
   snprintf((char *)&out_file[0], 256, outfile_pattern, num);
   
   if((*fdin = open((const char*)&in_file[0], O_RDONLY, 0644)) < 0)
   {
      printf("Error opening %s\n", in_file);
      return false;
   }
   
   if((*fdout = open((const char*)&out_file[0], (O_RDWR | O_CREAT), 0666)) < 0)
   {
      printf("Error opening %s\n", out_file);
      return false;
   }

   return true;
}

#ifdef USE_CUDA

/* Our main cuda kernel */
__global__ void cudaKernel (UINT8 *RR, UINT8 *GG, UINT8 *BB,
                            int NN)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   
   if(idx < NN)
   {
      RR[idx] = (0.30 * RR[idx]) + (0.59 * GG[idx]) + (0.11 * BB[idx]);
      GG[idx] = RR[idx];
      BB[idx] = RR[idx];
   }
}

void transform_pixels (UINT8 *h_RR, UINT8 *h_GG, UINT8 *h_BB,
                       UINT8 *d_RR, UINT8 *d_GG, UINT8 *d_BB,
                       int NN)

{
   int block_size = 512;
   dim3 dimBlock(block_size);
   dim3 dimGrid(NN/block_size);

//   printf("block Size = %d, NN = %d\n", block_size, NN);
//   printf("Grid Size = %d\n",NN/block_size);

   hipMemcpy(d_RR, h_RR, NN, hipMemcpyHostToDevice);
   hipMemcpy(d_GG, h_GG, NN, hipMemcpyHostToDevice);
   hipMemcpy(d_BB, h_BB, NN, hipMemcpyHostToDevice);

   cudaKernel<<<dimGrid, dimBlock>>>(d_RR, d_GG, d_BB, NN);
   hipDeviceSynchronize();

   hipMemcpy(h_RR, d_RR, NN, hipMemcpyDeviceToHost);
   hipMemcpy(h_GG, d_GG, NN, hipMemcpyDeviceToHost);
   hipMemcpy(h_BB, d_BB, NN, hipMemcpyDeviceToHost);
}

#else

void convert_to_grayscale (UINT8 *Rin, UINT8 *Gin, UINT8 *Bin,
                           UINT8 *Rout, UINT8 *Gout, UINT8 *Bout,
                           int NN)
{
   int ii = 0;

   // Read RGB data
   for(ii = 0; ii < NN; ii++)
   {
      // Source: Wikipedia - http://en.wikipedia.org/wiki/Grayscale
      Rout[ii]=( 0.30 * Rin[ii] ) + ( 0.59 * Gin[ii] ) + ( 0.11 * Bin[ii] );
      Gout[ii]=Rout[ii];
      Bout[ii]=Rout[ii];
   }
}

void transform_pixels (UINT8 *Rin, UINT8 *Gin, UINT8 *Bin,
                       UINT8 *Rout, UINT8 *Gout, UINT8 *Bout,
                       int NN)
{
   convert_to_grayscale(Rin, Gin, Bin,
                        Rout, Gout, Bout,
                        NN);
}

#endif // USE_CUDA

void print_time_stats(int num_frames)
{
   int ii = 0;
   UINT64 totalTime = 0;

   if(0 == num_frames)
   {
      printf("No frames processed! Exiting!\n");
      exit (-1);
   }

   for(ii = 0; ii < num_frames; ii++)
   {
      totalTime += frame_times[ii];
   }

   printf("Total time taken to process %d frames: %llu mSecs\n", num_frames, totalTime);
   printf("Average time per frame: %llu mSecs\n", totalTime/num_frames);
}


#define NUM_ARGS (8)
int main(int argc, char *argv[])
{
   int fdin, fdout;
   int height = 0;
   int width = 0;
   int num_pixels = 0;
   int header_len = 0;
   int seq_start_num = 0;
   int seq_count = 0;
   int jj = 0;

   // Estimate CPU clock rate
   estimate_clk_rate();
    
   if(argc != NUM_ARGS)
   {
      printf("Usage: blacknwhite <infile%%d.ppm> <width> <height> <header_len> <outfile%%d.ppm> <seq_start_num> <count>\n");
      exit(-1);
   }
   else
   {
      width = atoi(argv[2]);
      height = atoi(argv[3]);
      header_len = atoi(argv[4]);
      seq_start_num = atoi(argv[6]);
      seq_count = atoi(argv[7]);

      num_pixels = width * height;

      printf("Using params: infile pattern: %s, outfile pattern: %s, \nheight: %d, width: %d, header_len: %d, seq_start: %d, seq_count: %d\n", 
             argv[1], argv[5], height, width, header_len, seq_start_num, seq_count);

      // Allocate memory for holding the pixels...
      header = (UINT8 *) malloc(header_len);
      h_R = (UINT8 *) malloc(num_pixels);
      h_G = (UINT8 *) malloc(num_pixels);
      h_B = (UINT8 *) malloc(num_pixels);

#ifdef USE_CUDA
      hipMalloc((void **) &d_R, num_pixels);
      hipMalloc((void **) &d_G, num_pixels);
      hipMalloc((void **) &d_B, num_pixels);
#else
      // Note: Even though these are named 'd_' for device memory,
      // In the case of NON CUDA code, we allocate them from the host.
      d_R = (UINT8 *) malloc(num_pixels);
      d_G = (UINT8 *) malloc(num_pixels);
      d_B = (UINT8 *) malloc(num_pixels);
#endif

      outfile = (UINT8 *) malloc(header_len + num_pixels*3);
      infile = (UINT8 *) malloc(header_len + num_pixels*3);

      // Allocate memory for computation.
      frame_times = (UINT8 *) malloc(seq_count);

      if(true != PARAMS_GOOD)
      {
         printf("Could not allocate the required memory!\n");
         exit(-1);
      }

      strncpy(infile_pattern, argv[1], sizeof(infile_pattern));
      strncpy(outfile_pattern, argv[5], sizeof(outfile_pattern));
   }

   for(jj=seq_start_num; jj<(seq_start_num + seq_count); jj++)
   {
      if( false == open_files(jj, &fdin, &fdout))
      {
         printf("open files failed! bailing out!\n");
         break;
      }

      read_input_from_file(fdin, num_pixels, header_len,
                           h_R, h_G, h_B);

/***************** Start of  core computation **************/
      save_start_time();
      transform_pixels(h_R, h_G, h_B,
                       d_R, d_G, d_B,
                       num_pixels);

      save_stop_time();
      frame_times[jj] = calc_time_diff();
/***************** End of core computation **************/

#ifdef USE_CUDA
      write_output_to_file(fdout, num_pixels, header_len,
                           h_R, h_G, h_B);
#else
      write_output_to_file(fdout, num_pixels, header_len,
                           d_R, d_G, d_B);
#endif

      close(fdin);
      close(fdout);

   } // Loop through sequence of images

   print_time_stats(jj);

   FREE_MEM;

   return 0;
}
